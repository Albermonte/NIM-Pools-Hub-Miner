#include "kernels.h"

__host__ void set_block_header(struct worker_t *worker, uint32_t threadIndex, nimiq_block_header *block_header)
{
    initial_seed inseed;
    inseed.lanes = 1;
    inseed.hash_len = ARGON2_HASH_LENGTH;
    inseed.memory_cost = NIMIQ_ARGON2_COST;
    inseed.iterations = 1;
    inseed.version = 0x13;
    inseed.type = 0;
    inseed.header_len = sizeof(nimiq_block_header);
    memcpy(&inseed.header, block_header, sizeof(nimiq_block_header));
    inseed.salt_len = NIMIQ_ARGON2_SALT_LEN;
    memcpy(&inseed.salt, NIMIQ_ARGON2_SALT, NIMIQ_ARGON2_SALT_LEN);
    inseed.secret_len = 0;
    inseed.extra_len = 0;
    memset(&inseed.padding, 0, sizeof(inseed.padding));
  
    hipMemcpyAsync(worker->inseed[threadIndex], &inseed, sizeof(initial_seed), hipMemcpyHostToDevice);
    hipMemsetAsync(worker->nonce[threadIndex], 0, sizeof(uint32_t)); // zero nonce
}

__host__ hipError_t mine_nonces(struct worker_t *worker, uint32_t threadIndex, uint32_t start_nonce, uint32_t share_compact, uint32_t *nonce)
{
    init_memory<<<worker->init_memory_blocks, worker->init_memory_threads>>>(worker->memory[threadIndex], worker->inseed[threadIndex], start_nonce);
    argon2<<<worker->argon2_blocks, worker->argon2_threads, worker->cacheSize * ARGON2_BLOCK_SIZE>>>(worker->memory[threadIndex], worker->cacheSize, worker->memoryTradeoff);
    get_nonce<<<worker->get_nonce_blocks, worker->get_nonce_threads>>>(worker->memory[threadIndex], start_nonce, share_compact, worker->nonce[threadIndex]);

    hipError_t result = hipStreamSynchronize(0);
    if (result != hipSuccess)
    {
        return result;
    }

    hipMemcpy(nonce, worker->nonce[threadIndex], sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (*nonce > 0)
    {
        hipMemsetAsync(worker->nonce[threadIndex], 0, sizeof(uint32_t)); // zero nonce
    }

    return hipSuccess;
}
