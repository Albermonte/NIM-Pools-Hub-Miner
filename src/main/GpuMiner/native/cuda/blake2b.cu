#include "hip/hip_runtime.h"
#include "kernels.h"


#define IV0 0x6a09e667f3bcc908UL
#define IV1 0xbb67ae8584caa73bUL
#define IV2 0x3c6ef372fe94f82bUL
#define IV3 0xa54ff53a5f1d36f1UL
#define IV4 0x510e527fade682d1UL
#define IV5 0x9b05688c2b3e6c1fUL
#define IV6 0x1f83d9abfb41bd6bUL
#define IV7 0x5be0cd19137e2179UL

__constant__ static const uint8_t sigma[12][16] = {
    {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
    {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
    {11, 8, 12, 0, 5, 2, 15, 13, 10, 14, 3, 6, 7, 1, 9, 4},
    {7, 9, 3, 1, 13, 12, 11, 14, 2, 6, 5, 10, 4, 0, 15, 8},
    {9, 0, 5, 7, 2, 4, 10, 15, 14, 1, 11, 12, 6, 8, 3, 13},
    {2, 12, 6, 10, 0, 11, 8, 3, 4, 13, 7, 5, 15, 14, 1, 9},
    {12, 5, 1, 15, 14, 13, 4, 10, 0, 7, 6, 3, 9, 2, 8, 11},
    {13, 11, 7, 14, 12, 1, 3, 9, 5, 0, 15, 4, 8, 6, 2, 10},
    {6, 15, 14, 9, 11, 3, 0, 8, 12, 2, 13, 7, 1, 4, 10, 5},
    {10, 2, 8, 4, 7, 6, 1, 5, 15, 11, 9, 14, 3, 12, 13, 0},
    {0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15},
    {14, 10, 4, 8, 9, 15, 13, 6, 1, 12, 0, 2, 11, 7, 5, 3},
};

__device__ __forceinline__ uint64_t rotr64(uint64_t x, uint32_t n)
{
    return (x >> n) | (x << (64 - n));
}

__device__ __forceinline__ uint64_t swap64(uint64_t x)
{
    return ((x & 0x00000000000000FFUL) << 56)
        | ((x & 0x000000000000FF00UL) << 40)
        | ((x & 0x0000000000FF0000UL) << 24)
        | ((x & 0x00000000FF000000UL) <<  8)
        | ((x & 0x000000FF00000000UL) >>  8)
        | ((x & 0x0000FF0000000000UL) >> 24)
        | ((x & 0x00FF000000000000UL) >> 40)
        | ((x & 0xFF00000000000000UL) >> 56);
}

__device__ void blake2b_init(uint64_t *h, uint32_t hashlen)
{
    h[0] = IV0 ^ (0x01010000 | hashlen);
    h[1] = IV1;
    h[2] = IV2;
    h[3] = IV3;
    h[4] = IV4;
    h[5] = IV5;
    h[6] = IV6;
    h[7] = IV7;
}

__device__ void g(uint64_t *a, uint64_t *b, uint64_t *c, uint64_t *d, uint64_t m1, uint64_t m2)
{
    asm("{"
        ".reg .u64 s, x;"
        ".reg .u32 l1, l2, h1, h2;"
        // a = a + b + x
        "add.u64 %0, %0, %1;"
        "add.u64 %0, %0, %4;"
        // d = rotr64(d ^ a, 32)
        "xor.b64 x, %3, %0;"
        "mov.b64 {h1, l1}, x;"
        "mov.b64 %3, {l1, h1};"
        // c = c + d
        "add.u64 %2, %2, %3;"
        // b = rotr64(b ^ c, 24)
        "xor.b64 x, %1, %2;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x6543;"
        "prmt.b32 h2, l1, h1, 0x2107;"
        "mov.b64 %1, {l2, h2};"
        // a = a + b + y
        "add.u64 %0, %0, %1;"
        "add.u64 %0, %0, %5;"
        // d = rotr64(d ^ a, 16);
        "xor.b64 x, %3, %0;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x5432;"
        "prmt.b32 h2, l1, h1, 0x1076;"
        "mov.b64 %3, {l2, h2};"
        // c = c + d
        "add.u64 %2, %2, %3;"
        // b = rotr64(b ^ c, 63)
        "xor.b64 x, %1, %2;"
        "shl.b64 s, x, 1;"
        "shr.b64 x, x, 63;"
        "add.u64 %1, s, x;"
        "}"
        : "+l"(*a), "+l"(*b), "+l"(*c), "+l"(*d) : "l"(m1), "l"(m2)
    );
}

#define G(i, a, b, c, d) (g(&v[a], &v[b], &v[c], &v[d], m[sigma[r][2 * i]], m[sigma[r][2 * i + 1]]))

__device__ void blake2b_round(uint32_t r, uint64_t *v, uint64_t *m)
{
    G(0, 0, 4, 8, 12);
    G(1, 1, 5, 9, 13);
    G(2, 2, 6, 10, 14);
    G(3, 3, 7, 11, 15);
    G(4, 0, 5, 10, 15);
    G(5, 1, 6, 11, 12);
    G(6, 2, 7, 8, 13);
    G(7, 3, 4, 9, 14);
}

__device__ void blake2b_compress(uint64_t *h, uint64_t *m, uint32_t bytes_compressed, bool last_block)
{
    uint64_t v[BLAKE2B_QWORDS_IN_BLOCK];

    v[0] = h[0];
    v[1] = h[1];
    v[2] = h[2];
    v[3] = h[3];
    v[4] = h[4];
    v[5] = h[5];
    v[6] = h[6];
    v[7] = h[7];
    v[8] = IV0;
    v[9] = IV1;
    v[10] = IV2;
    v[11] = IV3;
    v[12] = IV4 ^ bytes_compressed;
    v[13] = IV5; // it's OK if below 2^32 bytes
    v[14] = last_block ? ~IV6 : IV6;
    v[15] = IV7;

    #pragma unroll
    for (uint32_t r = 0; r < 12; r++)
    {
        blake2b_round(r, v, m);
    }

    h[0] = h[0] ^ v[0] ^ v[8];
    h[1] = h[1] ^ v[1] ^ v[9];
    h[2] = h[2] ^ v[2] ^ v[10];
    h[3] = h[3] ^ v[3] ^ v[11];
    h[4] = h[4] ^ v[4] ^ v[12];
    h[5] = h[5] ^ v[5] ^ v[13];
    h[6] = h[6] ^ v[6] ^ v[14];
    h[7] = h[7] ^ v[7] ^ v[15];
}

__device__ __forceinline__ void set_nonce(uint64_t *buffer, uint32_t nonce)
{
    uint64_t n = (((uint64_t) __byte_perm(0, nonce, 0x0045)) << 32) | __byte_perm(0, nonce, 0x6700);
    buffer[5] = buffer[5] | n;
}

__device__ void compute_initial_hash(uint64_t *hash, uint64_t *inseed, uint32_t nonce)
{
    uint64_t buffer[BLAKE2B_QWORDS_IN_BLOCK];

    blake2b_init(hash, BLAKE2B_HASH_LENGTH);

    for (int i = 0; i < BLAKE2B_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = inseed[i];
    }
    blake2b_compress(hash, buffer, BLAKE2B_BLOCK_SIZE, false);

    for (int i = 0; i < BLAKE2B_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = inseed[BLAKE2B_QWORDS_IN_BLOCK + i];
    }
    set_nonce(buffer, nonce);
    blake2b_compress(hash, buffer, ARGON2_INITIAL_SEED_SIZE, true);
}

__device__ void fill_first_block(struct block_g *memory, uint64_t *inseed, uint32_t nonce, uint32_t block)
{
    uint64_t hash[8];
    compute_initial_hash(hash, inseed, nonce);

    uint32_t prehash_seed[BLAKE2B_BLOCK_SIZE / sizeof(uint32_t)];

    // Construct prehash seed
    prehash_seed[0] = ARGON2_BLOCK_SIZE;
    memcpy(&prehash_seed[1], hash, BLAKE2B_HASH_LENGTH);
    prehash_seed[17] = block;
    for (int i = 18; i < 32; i++)
    {
        prehash_seed[i] = 0;
    }

    ulonglong2 *dst = (ulonglong2*) memory->data;

    // V1
    blake2b_init(hash, BLAKE2B_HASH_LENGTH);
    blake2b_compress(hash, (uint64_t*) prehash_seed, ARGON2_PREHASH_SEED_SIZE, true);

    *(dst++) = *((ulonglong2*) &hash[0]);
    *(dst++) = *((ulonglong2*) &hash[2]);

    // V2-Vr
    uint64_t buffer[BLAKE2B_QWORDS_IN_BLOCK];
    for (int i = 8; i < BLAKE2B_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = 0;
    }

    for (int r = 2; r < 2 * ARGON2_BLOCK_SIZE / BLAKE2B_HASH_LENGTH; r++)
    {
        buffer[0] = hash[0];
        buffer[1] = hash[1];
        buffer[2] = hash[2];
        buffer[3] = hash[3];
        buffer[4] = hash[4];
        buffer[5] = hash[5];
        buffer[6] = hash[6];
        buffer[7] = hash[7];

        blake2b_init(hash, BLAKE2B_HASH_LENGTH);
        blake2b_compress(hash, buffer, BLAKE2B_HASH_LENGTH, true);

        *(dst++) = *((ulonglong2*) &hash[0]);
        *(dst++) = *((ulonglong2*) &hash[2]);
    }

    *(dst++) = *((ulonglong2*) &hash[4]);
    *(dst++) = *((ulonglong2*) &hash[6]);
}

__device__ void compact_to_target(uint32_t share_compact, uint64_t *target)
{
    uint32_t offset = (share_compact >> 24) - 3; // offset in bytes
    uint64_t value = share_compact & 0xFFFFFF;
    uint64_t hi = value >> ((8 - offset & 0x7) << 3);
    uint64_t lo = value << ((offset & 0x7) << 3); // value << (8 * (offset % 8))

    target[0] = (offset >= 24) ? lo : (offset > 20) ? hi : 0;
    target[1] = (offset >= 24) ? 0 : (offset >= 16) ? lo : (offset > 12) ? hi : 0;
    target[2] = (offset >= 16) ? 0 : (offset >= 8) ? lo : (offset > 4) ? hi : 0;
    target[3] = (offset < 8) ? lo : 0;
}

__device__ bool is_proof_of_work(uint64_t *hash, uint64_t *target)
{
    #pragma unroll
    for (uint32_t i = 0; i < 4; i++)
    {
        if (swap64(hash[i]) < target[i]) return true;
        if (swap64(hash[i]) > target[i]) return false;
    }
    return true;
}

__device__ void hash_last_block(struct block_g *memory, uint64_t *hash)
{
    uint64_t buffer[BLAKE2B_QWORDS_IN_BLOCK];
    uint32_t hi, lo;
    uint32_t bytes_compressed = 0;
    uint32_t bytes_remaining = ARGON2_BLOCK_SIZE;
    uint32_t *src = (uint32_t *)memory->data;

    blake2b_init(hash, ARGON2_HASH_LENGTH);

    hi = *(src++);
    buffer[0] = ARGON2_HASH_LENGTH | ((uint64_t)hi << 32);

    #pragma unroll
    for (uint32_t i = 1; i < BLAKE2B_QWORDS_IN_BLOCK; i++)
    {
        lo = *(src++);
        hi = *(src++);
        buffer[i] = lo | ((uint64_t)hi << 32);
    }

    bytes_compressed += BLAKE2B_BLOCK_SIZE;
    bytes_remaining -= (BLAKE2B_BLOCK_SIZE - sizeof(uint32_t));
    blake2b_compress(hash, buffer, bytes_compressed, false);

    while (bytes_remaining > BLAKE2B_BLOCK_SIZE)
    {
        #pragma unroll
        for (uint32_t i = 0; i < BLAKE2B_QWORDS_IN_BLOCK; i++)
        {
            lo = *(src++);
            hi = *(src++);
            buffer[i] = lo | ((uint64_t)hi << 32);
        }
        bytes_compressed += BLAKE2B_BLOCK_SIZE;
        bytes_remaining -= BLAKE2B_BLOCK_SIZE;
        blake2b_compress(hash, buffer, bytes_compressed, false);
    }

    buffer[0] = *src;
    #pragma unroll
    for (uint32_t i = 1; i < BLAKE2B_QWORDS_IN_BLOCK; i++)
    {
        buffer[i] = 0;
    }
    bytes_compressed += bytes_remaining;
    blake2b_compress(hash, buffer, bytes_compressed, true);
}

__global__ void init_memory(struct block_g *memory, uint64_t *inseed, uint32_t start_nonce)
{
    uint32_t job_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t nonce = start_nonce + job_id;
    uint32_t block = threadIdx.y;

    memory += (size_t)job_id * MEMORY_COST + block;
    fill_first_block(memory, inseed, nonce, block);
}

__global__ void get_nonce(struct block_g *memory, uint32_t start_nonce, uint32_t share_compact, uint32_t *nonce)
{
    uint32_t job_id = blockIdx.x * blockDim.x + threadIdx.x;
    memory += (size_t)(job_id + 1) * MEMORY_COST - 1;

    uint64_t hash[8];
    uint64_t target[4];

    compact_to_target(share_compact, target);
    hash_last_block(memory, hash);

    if (is_proof_of_work(hash, target))
    {
        atomicCAS(nonce, 0, start_nonce + job_id);
    }
}
