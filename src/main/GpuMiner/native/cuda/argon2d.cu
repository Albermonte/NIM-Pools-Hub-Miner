#include "hip/hip_runtime.h"
/*
MIT License

Copyright (c) 2016 Ondrej Mosnáček

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*/

/*
* Argon2d
* Simplified and refactored version of https://gitlab.com/omos/argon2-gpu
*/

#include "kernels.h"

struct block_th
{
    uint64_t a, b, c, d;
};

__device__ void move_block(struct block_th *dst, const struct block_th *src)
{
    *dst = *src;
}

__device__ void xor_block(struct block_th *dst, const struct block_th *src)
{
    dst->a ^= src->a;
    dst->b ^= src->b;
    dst->c ^= src->c;
    dst->d ^= src->d;
}

__device__ void load_block_cache(struct block_th *dst, const struct block_g *src, uint32_t thread)
{
    dst->a = src->data[0 * THREADS_PER_LANE + thread];
    dst->b = src->data[1 * THREADS_PER_LANE + thread];
    dst->c = src->data[2 * THREADS_PER_LANE + thread];
    dst->d = src->data[3 * THREADS_PER_LANE + thread];
}

__device__ void load_block_global(struct block_th *dst, const struct block_g *src, uint32_t thread)
{
    ulonglong2 *u128 = (ulonglong2*) src->data;
    asm("ld.global.ca.v2.u64 {%0, %1}, [%2];" : "=l"(dst->a), "=l"(dst->b) : "l"(&u128[0 * THREADS_PER_LANE + thread]));
    asm("ld.global.ca.v2.u64 {%0, %1}, [%2];" : "=l"(dst->c), "=l"(dst->d) : "l"(&u128[1 * THREADS_PER_LANE + thread]));
}

__device__ void store_block_cache(struct block_g *dst, const struct block_th *src, uint32_t thread)
{
    dst->data[0 * THREADS_PER_LANE + thread] = src->a;
    dst->data[1 * THREADS_PER_LANE + thread] = src->b;
    dst->data[2 * THREADS_PER_LANE + thread] = src->c;
    dst->data[3 * THREADS_PER_LANE + thread] = src->d;
}

__device__ void store_block_global(struct block_g *dst, const struct block_th *src, uint32_t thread)
{
    asm("st.global.wb.v2.u64 [%0], {%1, %2};" :: "l"(&dst->data[0 * THREADS_PER_LANE + 2 * thread]), "l"(src->a), "l"(src->b));
    asm("st.global.wb.v2.u64 [%0], {%1, %2};" :: "l"(&dst->data[2 * THREADS_PER_LANE + 2 * thread]), "l"(src->c), "l"(src->d));
}

__device__ void g(struct block_th *block)
{
    asm("{"
        ".reg .u64 s, x;"
        ".reg .u32 l1, l2, h1, h2;"
        // a = f(a, b);
        "add.u64 s, %0, %1;"            // s = a + b
        "cvt.u32.u64 l1, %0;"           // xlo = u64_lo(a)
        "cvt.u32.u64 l2, %1;"           // ylo = u64_lo(b)
        "mul.hi.u32 h1, l1, l2;"        // umulhi(xlo, ylo)
        "mul.lo.u32 l1, l1, l2;"        // xlo * ylo
        "mov.b64 x, {l1, h1};"          // x = u64_build(umulhi(xlo, ylo), xlo * ylo)
        "shl.b64 x, x, 1;"              // x = 2 * x
        "add.u64 %0, s, x;"             // a = s + x
        // d = rotr64(d ^ a, 32);
        "xor.b64 x, %3, %0;"
        "mov.b64 {h2, l2}, x;"
        "mov.b64 %3, {l2, h2};"         // swap hi and lo = rotr64(x, 32)
        // c = f(c, d);
        "add.u64 s, %2, %3;"
        "cvt.u32.u64 l1, %2;"
        "mul.hi.u32 h1, l1, l2;"
        "mul.lo.u32 l1, l1, l2;"
        "mov.b64 x, {l1, h1};"
        "shl.b64 x, x, 1;"
        "add.u64 %2, s, x;"
        // b = rotr64(b ^ c, 24);
        "xor.b64 x, %1, %2;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x6543;"  // permute bytes 76543210 => 21076543
        "prmt.b32 h2, l1, h1, 0x2107;"  // rotr64(x, 24)
        "mov.b64 %1, {l2, h2};"
        // a = f(a, b);
        "add.u64 s, %0, %1;"
        "cvt.u32.u64 l1, %0;"
        "mul.hi.u32 h1, l1, l2;"
        "mul.lo.u32 l1, l1, l2;"
        "mov.b64 x, {l1, h1};"
        "shl.b64 x, x, 1;"
        "add.u64 %0, s, x;"
        // d = rotr64(d ^ a, 16);
        "xor.b64 x, %3, %0;"
        "mov.b64 {l1, h1}, x;"
        "prmt.b32 l2, l1, h1, 0x5432;"  // permute bytes 76543210 => 10765432
        "prmt.b32 h2, l1, h1, 0x1076;"  // rotr64(x, 16)
        "mov.b64 %3, {l2, h2};"
        // c = f(c, d);
        "add.u64 s, %2, %3;"
        "cvt.u32.u64 l1, %2;"
        "mul.hi.u32 h1, l1, l2;"
        "mul.lo.u32 l1, l1, l2;"
        "mov.b64 x, {l1, h1};"
        "shl.b64 x, x, 1;"
        "add.u64 %2, s, x;"
        // b = rotr64(b ^ c, 63);
        "xor.b64 x, %1, %2;"
        "shl.b64 s, x, 1;"              // x << 1
        "shr.b64 x, x, 63;"             // x >> 63
        "add.u64 %1, s, x;"             // emits less instructions than "or"
        "}"
        : "+l"(block->a), "+l"(block->b), "+l"(block->c), "+l"(block->d)
    );
}

__device__ void transpose1(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr = (thread ^ 0x2);
    uint32_t g2 = (thread & 0x2);
    uint32_t g4 = (thread & 0x4);

    uint64_t xab = __shfl_sync(0xFFFFFFFF, g2 ? block->a : block->b, src_thr);
    uint64_t xcd = __shfl_sync(0xFFFFFFFF, g2 ? block->c : block->d, src_thr);

    uint64_t xa = g2 ? xab : block->a;
    uint64_t xc = g2 ? xcd : block->c;
    uint64_t xac = __shfl_xor_sync(0xFFFFFFFF, g4 ? xa : xc, 0x4);

    uint64_t xb = g2 ? block->b : xab;
    uint64_t xd = g2 ? block->d : xcd;
    uint64_t xbd = __shfl_xor_sync(0xFFFFFFFF, g4 ? xb : xd, 0x4);

    block->a = g4 ? xac : xa;
    block->b = g4 ? xbd : xb;
    block->c = g4 ? xc : xac;
    block->d = g4 ? xd : xbd;
}

__device__ void transpose2(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr = (thread ^ 0x10);
    uint32_t g4 = (thread & 0x4);
    uint32_t g16 = (thread & 0x10);

    uint64_t xac = __shfl_xor_sync(0xFFFFFFFF, g4 ? block->a : block->c, 0x4);
    uint64_t xbd = __shfl_xor_sync(0xFFFFFFFF, g4 ? block->b : block->d, 0x4);

    uint64_t xa = g4 ? xac : block->a;
    uint64_t xb = g4 ? xbd : block->b;
    uint64_t xab = __shfl_sync(0xFFFFFFFF, g16 ? xa : xb, src_thr);

    uint64_t xc = g4 ? block->c : xac;
    uint64_t xd = g4 ? block->d : xbd;
    uint64_t xcd = __shfl_sync(0xFFFFFFFF, g16 ? xc : xd, src_thr);

    block->a = g16 ? xab : xa;
    block->b = g16 ? xb : xab;
    block->c = g16 ? xcd : xc;
    block->d = g16 ? xd : xcd;
}

__device__ void transpose3(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr1 = (thread ^ 0x10);
    uint32_t src_thr2 = (thread ^ 0x2);
    uint32_t g2 = (thread & 0x2);
    uint32_t g16 = (thread & 0x10);

    uint64_t xab = __shfl_sync(0xFFFFFFFF, g16 ? block->a : block->b, src_thr1);
    uint64_t xcd = __shfl_sync(0xFFFFFFFF, g16 ? block->c : block->d, src_thr1);

    uint64_t xa = g16 ? xab : block->a;
    uint64_t xb = g16 ? block->b : xab;
    uint64_t xc = g16 ? xcd : block->c;
    uint64_t xd = g16 ? block->d : xcd;

    xab = __shfl_sync(0xFFFFFFFF, g2 ? xa : xb, src_thr2);
    xcd = __shfl_sync(0xFFFFFFFF, g2 ? xc : xd, src_thr2);

    block->a = g2 ? xab : xa;
    block->b = g2 ? xb : xab;
    block->c = g2 ? xcd : xc;
    block->d = g2 ? xd : xcd;
}

__device__ void shift1_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t mask = (thread & 0x2) >> 1;
    uint32_t src_thr_b = thread ^ mask ^ 0x2;
    uint32_t src_thr_d = thread ^ mask ^ 0x3;

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b, 0x4);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x1, 0x4);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d, 0x4);
}

__device__ void unshift1_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t mask = (thread & 0x2) >> 1;
    uint32_t src_thr_b = thread ^ mask ^ 0x3;
    uint32_t src_thr_d = thread ^ mask ^ 0x2;

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b, 0x4);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x1, 0x4);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d, 0x4);
}

__device__ void shift2_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr_b = thread ^ (((thread & 0x2) << 2) | 0x2);
    uint32_t src_thr_d = thread ^ (((~thread & 0x2) << 2) | 0x2);

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x8);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d);
}

__device__ void unshift2_shuffle(struct block_th *block, uint32_t thread)
{
    uint32_t src_thr_b = thread ^ (((~thread & 0x2) << 2) | 0x2);
    uint32_t src_thr_d = thread ^ (((thread & 0x2) << 2) | 0x2);

    block->b = __shfl_sync(0xFFFFFFFF, block->b, src_thr_b);
    block->c = __shfl_xor_sync(0xFFFFFFFF, block->c, 0x8);
    block->d = __shfl_sync(0xFFFFFFFF, block->d, src_thr_d);
}

__device__ void shuffle_block(struct block_th *block, uint32_t thread)
{
    transpose1(block, thread);

    g(block);

    shift1_shuffle(block, thread);

    g(block);

    unshift1_shuffle(block, thread);
    transpose2(block, thread);

    g(block);

    shift2_shuffle(block, thread);

    g(block);

    unshift2_shuffle(block, thread);
    transpose3(block, thread);
}

__device__ uint32_t compute_ref_index(struct block_th *prev, uint32_t curr_index)
{
    uint32_t ref_index = __shfl_sync(0xFFFFFFFF, (uint32_t) prev->a, 0);

    uint32_t ref_area_size = curr_index - 1;
    ref_index = __umulhi(ref_index, ref_index);
    ref_index = ref_area_size - 1 - __umulhi(ref_area_size, ref_index);
    return ref_index;
}

__global__ void argon2(struct block_g *memory, uint32_t cache_size, uint32_t memory_tradeoff)
{
    extern __shared__ struct block_g cache[];
    // ref_index of the current block, -1 if current block is stored to global mem
    __shared__ uint16_t ref_indexes[MEMORY_COST];

    uint32_t job_id = blockIdx.y;
    uint32_t thread = threadIdx.x;

    // select job's memory region
    memory += (size_t)job_id * MEMORY_COST;

    struct block_th prev_prev, prev, ref, tmp;
    bool is_stored = true;
    load_block_global(&tmp, memory, thread);
    load_block_global(&prev, memory + 1, thread);

    // cache first block
    store_block_cache(&cache[0], &tmp, thread);
    uint32_t curr_cache_pos = 1;

    ((uint64_t*) ref_indexes)[0 * THREADS_PER_LANE + thread] = (uint64_t) -1;
    ((uint64_t*) ref_indexes)[1 * THREADS_PER_LANE + thread] = (uint64_t) -1;
    ((uint64_t*) ref_indexes)[2 * THREADS_PER_LANE + thread] = (uint64_t) -1;
    ((uint64_t*) ref_indexes)[3 * THREADS_PER_LANE + thread] = (uint64_t) -1;

    for (uint32_t curr_index = 2; curr_index < MEMORY_COST; curr_index++)
    {
        move_block(&prev_prev, &prev);

        uint32_t ref_index = compute_ref_index(&prev, curr_index);
        uint32_t ref_ref_index = ref_indexes[ref_index];

        uint32_t ref_offset = curr_index - ref_index;
        if (ref_offset <= cache_size + 1)
        {
            uint32_t ref_cache_pos = curr_cache_pos + (cache_size + 1 - ref_offset);
            ref_cache_pos = (ref_cache_pos >= cache_size) ? ref_cache_pos - cache_size : ref_cache_pos;
            load_block_cache(&ref, &cache[ref_cache_pos], thread);
            xor_block(&prev, &ref);
        }
        else if (ref_ref_index == (uint16_t) -1)
        {
            load_block_global(&ref, memory + ref_index, thread);
            xor_block(&prev, &ref);
        }
        else
        {
            struct block_th ref_prev, ref_ref;

            load_block_global(&ref_prev, memory + ref_index - 1, thread);
            load_block_global(&ref_ref, memory + ref_ref_index, thread);
            xor_block(&ref_prev, &ref_ref);

            move_block(&tmp, &ref_prev);
            shuffle_block(&ref_prev, thread);
            xor_block(&ref_prev, &tmp);

            xor_block(&prev, &ref_prev);
        }

        move_block(&tmp, &prev);
        shuffle_block(&prev, thread);
        xor_block(&prev, &tmp);

        if (curr_index < MEMORY_COST - 1)
        {
            if (curr_index > 2 + cache_size
                && ref_indexes[curr_index - cache_size - 1] == (uint16_t) -1)
            {
                load_block_cache(&tmp, &cache[curr_cache_pos], thread);
                store_block_global(memory + curr_index - cache_size - 1, &tmp, thread);
            }

            store_block_cache(&cache[curr_cache_pos], &prev_prev, thread);

            is_stored = !is_stored || (curr_index < memory_tradeoff) || (ref_ref_index != (uint16_t) -1);
            if (!is_stored)
            {
                ref_indexes[curr_index] = ref_index;
            }

            curr_cache_pos++;
            curr_cache_pos = (curr_cache_pos == cache_size) ? 0 : curr_cache_pos;
        }
    }

    store_block_global(memory + MEMORY_COST - 1, &prev, thread);
}
